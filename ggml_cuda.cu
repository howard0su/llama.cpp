#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_fp16.h"

#include <assert.h>
#include <stdio.h>

#include "ggml.h"
#include "ggml-internal.h"

int cuda_device = -1; // -1: No CUDA, 0-N: one cuda card

int ggml_cuda_init(int prefered)
{
    int count;
    hipGetDeviceCount(&count);

    if (prefered < count)
        cuda_device = prefered;
    else
        cuda_device = 0;

    hipError_t cudaStatus = hipSetDevice(cuda_device);
    if (cudaStatus != hipSuccess) {
        return -1;
    }

    return 0;
}

void* ggml_cuda_allocate(int size)
{
    void *data;
    hipError_t cudaStatus = hipMalloc(&data, size);
    if (cudaStatus != hipSuccess) {
        GGML_PRINT("%s: not enough VRAM (needed %zu)\n",
            __func__, size);
        assert(false);
        return NULL;
    }

    return data;
}

void ggml_cuda_copy(void * target, const void * src, size_t size)
{
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(target, src, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        GGML_PRINT("%s: Failed to do the copy", __func__);
        assert(false);
    }
}

/////////////////////////////////////////////////////////////////////////////

// CUDA: use 512 threads per block
const int CAFFE_CUDA_NUM_THREADS = 512;

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}


template <typename Dtype>
__global__ void set_kernel(const int n, Dtype* y, const Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

////////////////////////////////////////////////////////////////////////////////

struct ggml_tensor * ggml_set_f32_cuda(struct ggml_tensor * tensor, float value) {
    const int n     = ggml_nrows(tensor);
    const int nc    = tensor->ne[0];
    const size_t n1 = tensor->nb[1];

    char * const data = (char * const)tensor->data;

    switch (tensor->type) {
        case GGML_TYPE_I8:
            {
                assert(tensor->nb[0] == sizeof(int8_t));
                for (int i = 0; i < n; i++) {
                    // ggml_vec_set_i8(nc, (int8_t *)(data + i*n1), value);
                    set_kernel<int8_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int8_t *)(data + i*n1), (int8_t)value);
                }
            } break;
        case GGML_TYPE_I16:
            {
                assert(tensor->nb[0] == sizeof(int16_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<int16_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int16_t *)(data + i*n1), (int16_t)value);
                }
            } break;
        case GGML_TYPE_I32:
            {
                assert(tensor->nb[0] == sizeof(int32_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<int32_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int32_t *)(data + i*n1), (int32_t)value);
                }
            } break;
        case GGML_TYPE_F16:
            {
                assert(tensor->nb[0] == sizeof(ggml_fp16_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<ggml_fp16_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (ggml_fp16_t *)(data + i*n1), (ggml_fp16_t)value);
                }
            } break;
        case GGML_TYPE_F32:
            {
                assert(tensor->nb[0] == sizeof(float));
                for (int i = 0; i < n; i++) {
                    set_kernel<float><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (float *)(data + i*n1), (float)value);
                }
            } break;
        default:
            {
                GGML_ASSERT(false);
            } break;
    }

    return tensor;
}

struct ggml_tensor * ggml_set_i32_cuda(struct ggml_tensor * tensor, int32_t value) {
    const int n     = ggml_nrows(tensor);
    const int nc    = tensor->ne[0];
    const size_t n1 = tensor->nb[1];

    char * const data = (char * const)tensor->data;

    switch (tensor->type) {
        case GGML_TYPE_I8:
            {
                assert(tensor->nb[0] == sizeof(int8_t));
                for (int i = 0; i < n; i++) {
                    // ggml_vec_set_i8(nc, (int8_t *)(data + i*n1), value);
                    set_kernel<int8_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int8_t *)(data + i*n1), (int8_t)value);
                }
            } break;
        case GGML_TYPE_I16:
            {
                assert(tensor->nb[0] == sizeof(int16_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<int16_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int16_t *)(data + i*n1), (int16_t)value);
                }
            } break;
        case GGML_TYPE_I32:
            {
                assert(tensor->nb[0] == sizeof(int32_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<int32_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (int32_t *)(data + i*n1), (int32_t)value);
                }
            } break;
        case GGML_TYPE_F16:
            {
                assert(tensor->nb[0] == sizeof(ggml_fp16_t));
                for (int i = 0; i < n; i++) {
                    set_kernel<ggml_fp16_t><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (ggml_fp16_t *)(data + i*n1), (ggml_fp16_t)value);
                }
            } break;
        case GGML_TYPE_F32:
            {
                assert(tensor->nb[0] == sizeof(float));
                for (int i = 0; i < n; i++) {
                    set_kernel<float><<<CAFFE_GET_BLOCKS(nc), CAFFE_CUDA_NUM_THREADS>>>(
                        nc, (float *)(data + i*n1), (float)value);
                }
            } break;
        default:
            {
                GGML_ASSERT(false);
            } break;
    }

    return tensor;
}


/////////////////////////////////////////////////////////////////////////////////////
static void ggml_compute_forward_get_rows_q(
        const struct ggml_compute_params * params,
        const struct ggml_tensor * src0,
        const struct ggml_tensor * src1,
              struct ggml_tensor * dst) {
}
static void ggml_compute_forward_get_rows_f16(
        const struct ggml_compute_params * params,
        const struct ggml_tensor * src0,
        const struct ggml_tensor * src1,
              struct ggml_tensor * dst) {
}
static void ggml_compute_forward_get_rows_f32(
        const struct ggml_compute_params * params,
        const struct ggml_tensor * src0,
        const struct ggml_tensor * src1,
              struct ggml_tensor * dst) {
}

static void ggml_compute_forward_get_rows(
        const struct ggml_compute_params * params,
        const struct ggml_tensor * src0,
        const struct ggml_tensor * src1,
        struct ggml_tensor * dst) {
    switch (src0->type) {
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q8_0:
            {
                ggml_compute_forward_get_rows_q(params, src0, src1, dst);
            } break;
        case GGML_TYPE_F16:
            {
                ggml_compute_forward_get_rows_f16(params, src0, src1, dst);
            } break;
        case GGML_TYPE_F32:
            {
                ggml_compute_forward_get_rows_f32(params, src0, src1, dst);
            } break;
        default:
            {
                GGML_ASSERT(false);
            } break;
    }
}


























///////////////////////////////////////////////////////////////////////////////////////
void ggml_compute_forward_cuda(struct ggml_compute_params * params, struct ggml_tensor * tensor) {
    GGML_ASSERT(params);

    switch (tensor->op) {
        case GGML_OP_GET_ROWS:
            {
                ggml_compute_forward_get_rows(params, tensor->src0, tensor->src1, tensor);
            } break;
        default:
            GGML_ASSERT(false);
    }
}